
#include <hip/hip_runtime.h>
#include <stdio.h>
#define N 512000
#define BLOCKSIZE 16
__global__ void add(int *a, int *b, int *c, int n) {
  int index = threadIdx.x + blockIdx.x * blockDim.x;
  if (index < n)
  c[index] = a[index] + b[index];
}

void random_ints(int *r, int n) {
  int i;
  for (i = 0; i < n; i++) {      
    r[i] = rand()%5000;
  }
}

int main(void) {

  int *a, *b, *c;
  // host copies of a, b, c
  int *d_a, *d_b, *d_c;
  // device copies of a, b, c
  int size = N * sizeof(int);

  // Allocate space for device copies of a, b, c
  hipMalloc((void **)&d_a, size);
  hipMalloc((void **)&d_b, size);
  hipMalloc((void **)&d_c, size);

  // Setup input values
  a = (int *)malloc(size); 
  random_ints(a, N);
  b = (int *)malloc(size); 
  random_ints(b, N);
  c = (int *)malloc(size);

  // Copy inputs to device
  hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
  hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);

  // Launch add() kernel on GPU
  add<<<(N + BLOCKSIZE - 1) / BLOCKSIZE, BLOCKSIZE>>>(d_a, d_b, d_c, N);

  // Copy result back to host
  hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);

  for (int i = 0; i < N; i++) {
    printf("(a, b, c) = (%d, %d, %d)\n", a[i], b[i], c[i]);
  }

  // Cleanup
  hipFree(d_a); hipFree(d_b); hipFree(d_c);
  free(a); free(b); free(c);


  return 0;
}
